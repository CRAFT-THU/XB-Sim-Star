#include "hip/hip_runtime.h"
#include "crossbar_cuda.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>

CROSSBAR entire_cb(1, ENTIRE_L, ENTIRE_W);

Crossbar::Crossbar() {}

Crossbar::~Crossbar() {
    hipFree(CB_cell);
    hipFree(std_d);
    delete []input;
    delete []output;
}

Crossbar::Crossbar(int n, int l, int w) {
    CB_n = n;
    CB_l = l;
    CB_w = w;
//    hipMalloc((void **)&CB_cell, CB_n * CB_l * CB_w * sizeof(float));
    CB_cell = new float[CB_n * CB_l * CB_w];
    input = new float[CB_n * CB_l * (AD_WIDTH/DA_WIDTH)];
    output = new float[CB_n * CB_w * (AD_WIDTH/DA_WIDTH)];
}

void Crossbar::init(){
    float* tmp_cell = new float[CB_n * CB_l * CB_w];
    // transform cb_cell
    for (int t = 0; t < CB_n; t++) {
        for (int i = 0; i < CB_w; i++) {
            for (int j = 0; j < CB_l; j++) {
                tmp_cell[t * CB_l * CB_w + i * CB_l + j] = CB_cell[t * CB_l * CB_w + j * CB_w + i];
            }
        }
    }

    delete []CB_cell;
    hipMalloc((void **)&CB_cell, CB_n * CB_l * CB_w * sizeof(float));

    hipMemcpy(CB_cell, tmp_cell, CB_n*CB_l*CB_w*sizeof(float), hipMemcpyHostToDevice);
    free(tmp_cell);
    get_std();
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    clock_t time;
    time = clock();
    hiprandSetPseudoRandomGeneratorSeed(gen, (int)time);
}

void Crossbar::run() {
    float *input_d, *output_d;
    hipMalloc((void **)&input_d, CB_n * CB_l * (AD_WIDTH/DA_WIDTH) * sizeof(float));
    hipMalloc((void **)&output_d, CB_n * CB_w * (AD_WIDTH/DA_WIDTH) * sizeof(float));
    hipMemcpy(input_d, input, CB_n * CB_l * (AD_WIDTH/DA_WIDTH) * sizeof(float), hipMemcpyHostToDevice);

//    dim3 numBlocks(CB_n, CB_l);
//    dim3 mul_numBlocks(CB_n, CB_w);
    // generate noise
//    float *temp_noise, *temp_cell, *temp_2;
//    hipMalloc((void **)&temp_noise, CB_n * CB_w * CB_l * sizeof(float));
//    hipMalloc((void **)&temp_cell, CB_n * CB_w * CB_l * sizeof(float));
//    hipMalloc((void **)&temp_2, CB_n * CB_w * CB_l * sizeof(float));
//    hiprandGenerateNormal(gen, temp_noise, CB_n*CB_l*CB_w, 0, 1);
//    CUDA_mmul<<<CB_l, CB_w>>>(temp_noise, std_d, temp_2);
//    CUDA_add<<<CB_l, CB_w>>>(CB_cell, temp_2, temp_cell);
//    hipMemcpy(CB_cell, temp_cell, CB_n * CB_l * CB_w * sizeof(float), hipMemcpyDeviceToDevice);

    // use cublas
    hipblasHandle_t handle;
    hipblasCreate(&handle);
//        float alpha = 1.0f, beta = 0.0f;
//        hipblasSgemv(handle, HIPBLAS_OP_T, CB_l, CB_w, &alpha, CB_cell, CB_l, input_d, 1, &beta, output_d, 1);
    float alpha = 1.0f, beta = 0.0f;
    int m = AD_WIDTH/DA_WIDTH, n = CB_w, k = CB_l;
//    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k,
//            &alpha, input_d, k, temp_cell, k, &beta, output_d, m);
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k,
                &alpha, input_d, k, CB_cell, k, &beta, output_d, m);
    hipblasDestroy(handle);

    hipMemcpy(output, output_d, CB_n * CB_w * (AD_WIDTH/DA_WIDTH) * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(input_d);
    hipFree(output_d);
//    hipFree(temp_noise);
//    hipFree(temp_cell);
//    hipFree(temp_2);
    // transpose output
    float *tmp_output = new float[CB_n * CB_w * (AD_WIDTH/DA_WIDTH)];
    for (int i = 0; i < AD_WIDTH / DA_WIDTH; ++i) {
        for (int j = 0; j < CB_w; ++j) {
            tmp_output[i*CB_w+j] = output[i + j * (AD_WIDTH/DA_WIDTH)];
        }
    }
    memcpy(output, tmp_output, sizeof(float)*CB_n * CB_w * (AD_WIDTH/DA_WIDTH));
    delete []tmp_output;
}

void Crossbar::init(float *CB_cells, int n, int l, int w)
{
    CB_l = l;
    CB_w = w;
    CB_n = n;
    hipMalloc((void **)&CB_cell, CB_n * CB_l * CB_w * sizeof(float));
    float* tmp_cell = new float[CB_n * CB_l * CB_w];
    // transform cb_cell
    for (int t = 0; t < CB_n; t++) {
        for (int i = 0; i < CB_w; i++) {
            for (int j = 0; j < CB_l; j++) {
                tmp_cell[t * CB_l * CB_w + i * CB_l + j] = CB_cells[t * CB_l * CB_w + j * CB_w + i];
            }
        }
    }

    hipMemcpy(CB_cell, tmp_cell, CB_n*CB_l*CB_w*sizeof(float), hipMemcpyHostToDevice);
    free(tmp_cell);
    get_std();
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    clock_t time;
    time = clock();
    hiprandSetPseudoRandomGeneratorSeed(gen, (int)time);
}

void Crossbar::printcrossbar() {
    float *temp_cell;
    temp_cell = (float*)calloc(CB_n * CB_l * CB_w, sizeof(float));
    //temp_cell = new float [CB_n*CB_l*CB_w];
    hipMemcpy(temp_cell, CB_cell, CB_n * CB_l * CB_w * sizeof(float), hipMemcpyDeviceToHost);
    printf ("_______________\n");
    for (int i = 0; i < CB_n; i++) {
        for (int j = 0; j < CB_l; j++) {
            for(int k = 0; k < CB_w; k++) {
                printf("%f,%d,%d,%d,%d ", temp_cell[i*CB_l*CB_w+j*CB_w+k], i, j, k, i*CB_l*CB_w+j*CB_w+k);
            }
            printf ("\n");
        }
        printf ("\n");
    }
    printf ("_______________\n");
    free(temp_cell);
}

void Crossbar::get_std() {//old formula -0.0006034 * (x * 1e3) ** 2 + 0.06184 * x + 0.7240 * 1e-6
    // new formula (-0.0006034 * (x * 40 + 4) ** 2 + 0.06184 * (x * 40 + 4) + 0.7240) * 0.025
//    dim3 numBlocks(CB_n, CB_l);
    hipMalloc((void **)&std_d, CB_n * CB_l * CB_w * sizeof(float));
    float *temp_1;
    hipMalloc((void **)&temp_1, CB_n * CB_l * CB_w * sizeof(float));
    //hipMemcpy(temp_1, CB_cell, CB_n*CB_l*CB_w* sizeof(float),hipMemcpyDeviceToDevice) ;
    CUDA_abs<<<CB_l, CB_w>>>(CB_cell, temp_1);

    float *temp_2;
    hipMalloc((void **)&temp_2, CB_n * CB_l * CB_w * sizeof(float));
    CUDA_mul<<<CB_l, CB_w>>>(temp_1, 40, temp_2);
    hipMemcpy(temp_1, temp_2, CB_n * CB_l * CB_w * sizeof(float), hipMemcpyDeviceToDevice);
    CUDA_shift<<<CB_l, CB_w>>>(temp_1, 4, temp_2);
    // temp_1 = temp_2 = x * 40 + 4
    hipMemcpy(temp_1, temp_2, CB_n * CB_l * CB_w * sizeof(float), hipMemcpyDeviceToDevice);

    float *temp_3;
    hipMalloc((void **)&temp_3, CB_n * CB_l * CB_w * sizeof(float));
    // temp_3 = (x * 40 + 4) ** 2
    CUDA_mmul<<<CB_l, CB_w>>>(temp_1, temp_2, temp_3);
    // temp_1 = -0.0006034 * (x * 40 + 4) ** 2
    CUDA_mul<<<CB_l, CB_w>>>(temp_3, -0.0006034, temp_1);
    // temp_3 = 0.06184 * (x * 40 + 4)
    CUDA_mul<<<CB_l, CB_w>>>(temp_2, 0.06184, temp_3);
    // temp_2 = -0.0006034 * (x * 40 + 4) ** 2 + 0.06184 * (x * 40 + 4)
    CUDA_add<<<CB_l, CB_w>>>(temp_1, temp_3, temp_2);
    // temp_1 = -0.0006034 * (x * 40 + 4) ** 2 + 0.06184 * (x * 40 + 4) + 0.7240
    CUDA_shift<<<CB_l, CB_w>>>(temp_2, 0.7240, temp_1);
    // temp_2 = (-0.0006034 * (x * 40 + 4) ** 2 + 0.06184 * (x * 40 + 4) + 0.7240) * 2.5
    CUDA_mul<<<CB_l, CB_w>>>(temp_1, 0.025, temp_2);

    hipMemcpy(std_d, temp_2, CB_n * CB_l * CB_w * sizeof(float), hipMemcpyDeviceToDevice);

    hipFree( temp_1 );
    hipFree( temp_2 );
    hipFree( temp_3 );
}

void Crossbar::get_noise(float *noise) {
    hipMalloc((void **)&noise, CB_n*CB_l*CB_w*sizeof(float));
    hiprandGenerateNormal(gen, noise, CB_n*CB_l*CB_w, 0, 1);
}

void Crossbar::printstd() {
    printf ("~~~~~~~~~~~~~~~~~~~\n");
    float *temp_cell;
    temp_cell = (float*)calloc(CB_n*CB_l*CB_w,sizeof(float));
    hipMemcpy(temp_cell, std_d, CB_n*CB_l*CB_w* sizeof(float),hipMemcpyDeviceToHost) ;
    for (int i=0;i<CB_n;i++)
    {
        for (int j=0;j<CB_l;j++)
        {
            for(int p=0;p<CB_w;p++)
            {
                printf("%f ",temp_cell[i*CB_l*CB_w+j*CB_w+p]);
            }
            printf ("\n");
        }
        printf ("\n");
    }
    printf ("~~~~~~~~~~~~~~~~~~~\n");
    free(temp_cell);
}

void Crossbar::run(float *input, float *output, bool use_noise) {
//    float *input_d, *output_d;
//    hipMalloc((void **)&input_d, CB_n * CB_l * sizeof(float));
//    hipMalloc((void **)&output_d, CB_n * CB_w * sizeof(float));
//    hipMemcpy(input_d, input, CB_n * CB_l * sizeof(float), hipMemcpyHostToDevice);
//
//    dim3 numBlocks(CB_n, CB_l);
//    dim3 mul_numBlocks(CB_n, CB_w);
//
//    if (use_noise) {
//        float *temp_noise, *temp_cell, *temp_2;
//        hipMalloc((void **)&temp_noise, CB_n * CB_w * CB_l * sizeof(float));
//        hipMalloc((void **)&temp_cell, CB_n * CB_w * CB_l * sizeof(float));
//        hipMalloc((void **)&temp_2, CB_n * CB_w * CB_l * sizeof(float));
//        get_noise(temp_noise);
//        CUDA_mmul<<<numBlocks, CB_w>>>(temp_noise, std_d, temp_2, CB_w, CB_l);
//        CUDA_add<<<numBlocks, CB_w>>>(CB_cell, temp_2, temp_cell, CB_w, CB_l);
//
//        CUDA_MatrixMui<<<mul_numBlocks, 1>>>(input_d, temp_cell, output_d, CB_l, CB_w);
//
//        hipFree(temp_noise);
//        hipFree(temp_cell);
//        hipFree(temp_2);
//    }
//    else {
//        CUDA_MatrixMui<<<mul_numBlocks, 1>>>(input_d, CB_cell, output_d, CB_l, CB_w);
//
//        // use cublas
////        hipblasHandle_t handle;
////        hipblasCreate(&handle);
////        float alpha = 1.0f, beta = 0.0f;
////        hipblasSgemv(handle, HIPBLAS_OP_T, CB_l, CB_w, &alpha, CB_cell, CB_l, input_d, 1, &beta, output_d, 1);
////        hipblasDestroy(handle);
//    }
//    hipMemcpy(output, output_d, CB_n * CB_w * sizeof(float), hipMemcpyDeviceToHost);
//    hipFree( input_d );
//    hipFree( output_d );
}

void Crossbar::free_space() {
    // do something
    free(CB_cell);
    free(std_d);
}
